#include "hip/hip_runtime.h"
/*
Copyright 2011. All rights reserved.
Institute of Measurement and Control Systems
Karlsruhe Institute of Technology, Germany

This file is part of libelas.
Authors: Andreas Geiger

libelas is free software; you can redistribute it and/or modify it under the
terms of the GNU General Public License as published by the Free Software
Foundation; either version 3 of the License, or any later version.

libelas is distributed in the hope that it will be useful, but WITHOUT ANY
WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
PARTICULAR PURPOSE. See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License along with
libelas; if not, write to the Free Software Foundation, Inc., 51 Franklin
Street, Fifth Floor, Boston, MA 02110-1301, USA 
*/
/*
Edited by: Mostafa A.Saleh
moustafa.i.saleh <at> gmail.com
*/
#include "../CPU/descriptor.h"
#include "../CPU/filter.h"
#include <emmintrin.h>

#include <iostream>

using namespace std;

__global__ void sobelKernel(const uint8_t* d_in, uint8_t* d_out_v, uint8_t* d_out_h, int w, int h)
{
  int32_t x = (blockIdx.x * blockDim.x) + threadIdx.x; //width
  int32_t y = (blockIdx.y * blockDim.y) + threadIdx.y; //height

  if (x < 3 || x > w || y > h || y < 3)//out of bounds check
    return;
    const int sobel_x[3][3] = {
        {-3, 0, 3},
        {-10, 0, 10},
        {-3, 0, 3}};
    const int sobel_y[3][3]  = {
        {-3,   -10,   -3},
        {0,   0,   0},
        {3,  10,  3}};
    int16_t magnitude_x = 0 ,magnitude_y = 0 ;
    for (int16_t j = -1; j <= 1; ++j) {
        for (int16_t i = -1; i <= 1; ++i) {
            magnitude_x += d_in[(y + j)*w +(x+i)] * sobel_x[j+1][i+1];
            magnitude_y += d_in[(y + j)*w +(x+i)] * sobel_y[j+1][i+1];
        }
    }
    //magnitude_y = (magnitude_y <0)?0:magnitude_y;
    //magnitude_x = (magnitude_x <0)?0:magnitude_x;
    magnitude_y = __sad(magnitude_y,0,0); //absolute value 
    magnitude_x = __sad(magnitude_x,0,0);//absolute value 
    magnitude_y = (magnitude_y >255)?255:magnitude_y;
    magnitude_x = (magnitude_x >255)?255:magnitude_x;
    d_out_v[y*w + x ] = magnitude_y;
    d_out_h[y*w + x] = magnitude_x; 
}
void sobelGPU( const uint8_t* in, uint8_t* out_v, uint8_t* out_h, int32_t w, int32_t h )
{
    uint8_t* d_in, *d_out_h, *d_out_v;
    hipMalloc((void**) &d_in, (w*h*sizeof(uint8_t))); //allocate input image in GPU
    hipMalloc((void**) &d_out_h, w*h*sizeof(uint8_t)); //allocate output x image in GPU
    hipMalloc((void**) &d_out_v, w*h*sizeof(uint8_t)); //allocate output y image in GPU
    hipMemcpy(d_in, in, w*h*sizeof(uint8_t), hipMemcpyHostToDevice); //copy input image to GPU
    dim3 threadsPerBlock(16,16,1);
    dim3 numBlocks( w/16, h/16,1); 
    sobelKernel<<<numBlocks , threadsPerBlock>>>(d_in,d_out_v,d_out_h,w,h);
    //hipDeviceSynchronize();
    hipMemcpy(out_h, d_out_h, w*h*sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(out_v, d_out_v, w*h*sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipFree(d_out_v);
    hipFree(d_out_h);
    hipFree(d_in);
}
Descriptor::Descriptor(uint8_t* I,int32_t width,int32_t height,int32_t bpl,bool half_resolution) {
  I_desc        = (uint8_t*)_mm_malloc(16*width*height*sizeof(uint8_t),16);
  uint8_t* I_du = (uint8_t*)_mm_malloc(bpl*height*sizeof(uint8_t),16);
  uint8_t* I_dv = (uint8_t*)_mm_malloc(bpl*height*sizeof(uint8_t),16);
  //Filter call so sobel filter to get lines better
 // filter::sobel3x3(I,I_du,I_dv,bpl,height);
   sobelGPU(I,I_du,I_dv,width,height);//fliped
  //Create 16 byte discriptors for each deep image pixel
 
  createDescriptor(I_du,I_dv,width,height,bpl,half_resolution);
  _mm_free(I_du);
  _mm_free(I_dv);
}

Descriptor::~Descriptor() {
  _mm_free(I_desc);
}

void Descriptor::createDescriptor (uint8_t* I_du,uint8_t* I_dv,int32_t width,int32_t height,int32_t bpl,bool half_resolution) {

  uint8_t *I_desc_curr;  
  uint32_t addr_v0,addr_v1,addr_v2,addr_v3,addr_v4;
  
  // do not compute every second line
  if (half_resolution) {
  
    // create filter strip
    for (int32_t v=4; v<height-3; v+=2) {

      addr_v2 = v*bpl; //Current line
      addr_v0 = addr_v2-2*bpl; //2 lines above
      addr_v1 = addr_v2-1*bpl; //1 lines above
      addr_v3 = addr_v2+1*bpl; //1 lines below
      addr_v4 = addr_v2+2*bpl; //2 lines below

      //Save the surrounding filtered rhombus point of interests (Total of 16 points)
      //Du is horizontal filter result
      //Dv is vertical filter result (more horizontal change in stero camera so we can use less vertical stuff)
      //du :
      // - - x - -
      // - x x x -
      // x x o x x
      // - x x x -
      // - - x - -
      //dv :
      // - - - - -
      // - - x - -
      // - x o x -
      // - - x - -
      // - - - - -
      for (int32_t u=3; u<width-3; u++) {
        I_desc_curr = I_desc+(v*width+u)*16;
        *(I_desc_curr++) = *(I_du+addr_v0+u+0);
        *(I_desc_curr++) = *(I_du+addr_v1+u-2);
        *(I_desc_curr++) = *(I_du+addr_v1+u+0);
        *(I_desc_curr++) = *(I_du+addr_v1+u+2);
        *(I_desc_curr++) = *(I_du+addr_v2+u-1);
        *(I_desc_curr++) = *(I_du+addr_v2+u+0);
        *(I_desc_curr++) = *(I_du+addr_v2+u+0);
        *(I_desc_curr++) = *(I_du+addr_v2+u+1);
        *(I_desc_curr++) = *(I_du+addr_v3+u-2);
        *(I_desc_curr++) = *(I_du+addr_v3+u+0);
        *(I_desc_curr++) = *(I_du+addr_v3+u+2);
        *(I_desc_curr++) = *(I_du+addr_v4+u+0);
        *(I_desc_curr++) = *(I_dv+addr_v1+u+0);
        *(I_desc_curr++) = *(I_dv+addr_v2+u-1);
        *(I_desc_curr++) = *(I_dv+addr_v2+u+1);
        *(I_desc_curr++) = *(I_dv+addr_v3+u+0);
      }
    }
    
  // compute full descriptor images
  } else {
    
    // create filter strip
    for (int32_t v=3; v<height-3; v++) {

      addr_v2 = v*bpl;
      addr_v0 = addr_v2-2*bpl;
      addr_v1 = addr_v2-1*bpl;
      addr_v3 = addr_v2+1*bpl;
      addr_v4 = addr_v2+2*bpl;

      for (int32_t u=3; u<width-3; u++) {
        I_desc_curr = I_desc+(v*width+u)*16;
        *(I_desc_curr++) = *(I_du+addr_v0+u+0);
        *(I_desc_curr++) = *(I_du+addr_v1+u-2);
        *(I_desc_curr++) = *(I_du+addr_v1+u+0);
        *(I_desc_curr++) = *(I_du+addr_v1+u+2);
        *(I_desc_curr++) = *(I_du+addr_v2+u-1);
        *(I_desc_curr++) = *(I_du+addr_v2+u+0);
        *(I_desc_curr++) = *(I_du+addr_v2+u+0);
        *(I_desc_curr++) = *(I_du+addr_v2+u+1);
        *(I_desc_curr++) = *(I_du+addr_v3+u-2);
        *(I_desc_curr++) = *(I_du+addr_v3+u+0);
        *(I_desc_curr++) = *(I_du+addr_v3+u+2);
        *(I_desc_curr++) = *(I_du+addr_v4+u+0);
        *(I_desc_curr++) = *(I_dv+addr_v1+u+0);
        *(I_desc_curr++) = *(I_dv+addr_v2+u-1);
        *(I_desc_curr++) = *(I_dv+addr_v2+u+1);
        *(I_desc_curr++) = *(I_dv+addr_v3+u+0);
      }
    }
  }
  
}

