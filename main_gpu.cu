
#include "opencv2/calib3d.hpp"
#include "opencv2/highgui.hpp"
#include <opencv2/imgproc.hpp>
#include <opencv2/contrib/contrib.hpp>
#include <opencv2/core/core.hpp>

#include <iostream>
#include "CPU/elas.h"
#include "GPU/elas_gpu.h"
#include "CPU/image.h"

using namespace std;
using namespace cv;
// Enable profiling
//#define PROFILE

int main(int argc, char** argv) {

  // Startup the GPU device
  // https://devtalk.nvidia.com/default/topic/895513/cuda-programming-and-performance/cudamalloc-slow/post/4724457/#4724457
  hipFree(0);
  Mat colormap; //V the concatenated images , Right_color the is the 3 channel origianl right frame , used in viewing 
  Elas::parameters param;
 param.postprocess_only_left = false;
 ElasGPU elas(param);
 cv::Mat dst;
  Mat leftim=imread("I1_000606.png",CV_LOAD_IMAGE_GRAYSCALE);
  Mat rightim=imread("I2_000606.png",CV_LOAD_IMAGE_GRAYSCALE);
  
  // get image width and height
  int32_t width  = leftim.cols;
  int32_t height = leftim.rows;
  const int32_t dims[3] = {width,height,width}; // bytes per line = width
  // allocate memory for disparity images
  float* D1_data = (float*)malloc(width*height*sizeof(float));
  float* D2_data = (float*)malloc(width*height*sizeof(float));
	cout<<"HERE\n";
  elas.process(leftim.data,rightim.data,D1_data,D2_data,dims);

  Mat L1(height, width,CV_32FC1,D1_data);
  normalize(L1, colormap, 0, 255, NORM_MINMAX, CV_8U); //to view it
  applyColorMap(colormap, colormap, COLORMAP_JET);  //to make it colored
  adaptiveBilateralFilter(colormap,dst, cv::Size(11, 11), 50 );

  //applyColorMap(L1, colormap, COLORMAP_JET);  //to make it colored
	 // imshow("disp",colormap);
    imwrite("Disparityn.png",dst);
    imwrite("Disparity.png",L1);
  //waitKey(0);
  free(D1_data);
  free(D2_data);


  return 0;
}
