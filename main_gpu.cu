
#include "opencv2/calib3d.hpp"
#include "opencv2/highgui.hpp"
#include <opencv2/imgproc.hpp>
#include <opencv2/contrib/contrib.hpp>
#include <opencv2/core/core.hpp>

#include <iostream>
#include "elas.h"
#include "GPU/elas_gpu.h"
#include "CPU/image.h"

using namespace std;
using namespace cv;
// Enable profiling
//#define PROFILE

int main(int argc, char** argv) {

  // Startup the GPU device
  // https://devtalk.nvidia.com/default/topic/895513/cuda-programming-and-performance/cudamalloc-slow/post/4724457/#4724457
  hipFree(0);
  Mat colormap; //V the concatenated images , Right_color the is the 3 channel origianl right frame , used in viewing 
  Elas::parameters param;
 param.postprocess_only_left = false;
 ElasGPU elas(param);

  Mat leftim=imread("input/cones_left.pgm",CV_LOAD_IMAGE_GRAYSCALE);
  Mat rightim=imread("input/cones_right.pgm",CV_LOAD_IMAGE_GRAYSCALE);
  
  // get image width and height
  int32_t width  = leftim.cols;
  int32_t height = leftim.rows;
  const int32_t dims[3] = {width,height,width}; // bytes per line = width
  // allocate memory for disparity images
  float* D1_data = (float*)malloc(width*height*sizeof(float));
  float* D2_data = (float*)malloc(width*height*sizeof(float));
	cout<<"HERE\n";
  elas.process(leftim.data,rightim.data,D1_data,D2_data,dims);

  Mat L1(height, width,CV_32FC1,D1_data);
  Mat R(height, width,CV_32FC1,D2_data);
  normalize(L1, L1, 0, 255, NORM_MINMAX, CV_8U); //to view it
  applyColorMap(L1, colormap, COLORMAP_JET);  //to make it colored
	  imshow("disp",colormap);
  waitKey(0);
  free(D1_data);
  free(D2_data);


  return 0;
}